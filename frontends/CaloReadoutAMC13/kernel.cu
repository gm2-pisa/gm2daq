#include "hip/hip_runtime.h"
/**
 * @file    kernel.cu
 * @author  Vladimir Tishchenko <tishenko@pa.uky.edu>
 * @date    Last-Updated: Tue Nov 24 13:32:35 2015 (-0600)
 *          By : Data Acquisition
 *          Update #: 710 
 * @version $Id$
 * @copyright (c) new (g-2) collaboration
 *
 * 
 * 
 * @section Changelog
 * @verbatim
 * $Log$
 * @endverbatim
 */

#include <stdlib.h>
#include <stdio.h>
#include <stdint.h>
#include <string.h>
#include <math.h>
#include <linux/types.h>
//#include <cuPrintf.cu>

#ifdef DEBUG
#define dbprintf(...) printf(__VA_ARGS__)
#else
#define dbprintf(...) 
#endif

// includes, project
#include <hip/hip_runtime.h>
//#include "cuPrintf.cu"
#include "cuda_tools_g2.h"
#include "gpu_thread.h"
#include "midas.h"
#include "amc13_odb.h"

// number of detectors in calorimeter
#define N_DETECTORS    54
#define N_SEGMENTS_X   9 
#define N_SEGMENTS_Y   6
// number of samples per waveform
//#define N_SAMPLES      368640 // 500 MSPS
//#define N_SAMPLES       589824  //800 MSPS
//#define N_SAMPLES       256  //TEST
//#define N_SAMPLES     233005  // for testing - nearest number of samples for AMC13 event generator 0x3fff0 payload divided by 54 and rou
// no. of channels per Rider

// introduced N_SAMPLES_MAX for use in definition of structures GPU_HIS_DATA, 
// GPU_AUX_DATA that are mapped to regions of gpu_odata that containing the hitogram data 
// and auxiliary data to avoid the need to use an ODB parameter in these structures
#define N_SAMPLES_MAX 589824

#define USE_RIDER_FORMAT  1
#define N_RIDERCHANS    5 

// ADC type
#define ADC_TYPE       int16_t
#define ADC_MAX        2048
//#define DECIMATION     32

// structure for auxiliary data
typedef struct s_gpu_his_data {

   int16_t  wf_hist[N_SAMPLES_MAX*N_DETECTORS];  // sum waveform , int array of size N_SAMPLES_MAX*N_DETECTORS
}  GPU_HIS_DATA;

// structure for auxiliary data
typedef struct s_gpu_aux_data {
  
  double   wf_sum[N_SAMPLES_MAX];        // sum waveform , double array of size N_SAMPLES_MAX
  double   pedestal[N_DETECTORS];        // calculated pedestal average,  double array of size N_DETECTORS
  int      island_pattern[N_SAMPLES_MAX];// auxiliary array for island build
  int      islands_size;                 // total size of the array islands[]
  struct {
    int time;
    int length;
    int offset;
  } island_info[N_SAMPLES_MAX]; // structure array of size N_SAMPLES_MAX
} GPU_AUX_DATA;

// structure for output data
typedef struct s_gpu_out_data {
  int island_offset;                // used to record islands
  int n_islands;                    // number of islands found
  int CTAG ;                        //number of islands>2 GeV && t>50us
  int16_t islands[1];                   // array of islands
} GPU_OUT_DATA;

// energy calibration coefficient - in the future this needs 
// to be recorded to ODB detectors may require recalibration from time to time
__device__
static double A_calib[N_DETECTORS] = {
  1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0,
  1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0,
  1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0,
  1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0,
  1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0,
  1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0,
  1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0,
  1.0, 1.0, 1.0, 1.0, 1.0
};

__constant__ int DEV_N_samples; 
int HOST_N_samples;
__constant__ int DEV_N_presamples;
int HOST_N_presamples;
__constant__ int DEV_N_postsamples;
int HOST_N_postsamples;
__constant__ int DEV_decimation; 
int HOST_decimation;
__constant__ bool DEV_threshold_sign;
bool HOST_threshold_sign;
__constant__ int DEV_threshold;
int HOST_threshold;
__constant__ int DEV_pedestal_option;
int HOST_pedestal_option;
__constant__ int DEV_global_pedestal;
int HOST_global_pedestal;

__constant__ int DEV_first_sample_index[N_DETECTORS]; 
int HOST_first_sample_index[N_DETECTORS];
__constant__ int DEV_island_option;
int HOST_island_option;

// arrays for mapping from segment identifier (x,y) to Rider module, channel
__constant__ int SegXYtoRiderModu[N_SEGMENTS_X][N_SEGMENTS_Y];
__constant__ int SegXYtoRiderChan[N_SEGMENTS_X][N_SEGMENTS_Y];

// arrays for testing  SegXYtoRiderModu[][], SegXYtoRiderChan[][]
int HostSegXYtoRiderModu[N_SEGMENTS_X][N_SEGMENTS_Y]= {
  1,  1,  1,  1,  1,  2,
  2,  2,  2,  2,  3,  3,
  3,  3,  3,  4,  4,  4,
  4,  4,  5,  5,  5,  5,
  5,  6,  6,  6,  6,  6,
  7,  7,  7,  7,  7,  8,
  8,  8,  8,  8,  9,  9,
  9,  9,  9,  10, 10, 10,
  10, 10, 11, 11, 11, 11,
};
int HostSegXYtoRiderChan[N_SEGMENTS_X][N_SEGMENTS_Y]= {
  1, 2, 3, 4, 5, 1,
  2, 3, 4, 5, 1, 2,
  3, 4, 5, 1, 2, 3,
  4, 5, 1, 2, 3, 4,
  5, 1, 2, 3, 4, 5,
  1, 2, 3, 4, 5, 1,
  2, 3, 4, 5, 1, 2,
  3, 4, 5, 1, 2, 3,
  4, 5, 1, 2, 3, 4
};

/*
// testing structure for Rider parameters
typedef struct s_riderparams {
  int nrmhwords;
  int nrmtwords;
  int nrchwords;
  int nrctwords;
} RIDERPARAMS;

RIDERPARAMS *HostRiderParams;
__constant__ RIDERPARAMS RiderParams;
*/

#if 0
/** 
 * Makes the distribution of ADC samples (fills ADC arrays
 * in GPU_AUX_DATA)
 * Histogramming is a bad task for GPU
 * Need a better solution
 * 
 * @param gpu_idata 
 * @param gpu_odata 
 */
__global__
// kernel_wf_make_ADC is no longer used
void kernel_wf_make_ADC(ADC_TYPE *gpu_idata, ADC_TYPE* gpu_odata)
{

  // access thread id
  const unsigned int tid = threadIdx.x;
  // access number of threads in this block
  const unsigned int num_threads = blockDim.x;
  // access block id
  const unsigned int bid = blockIdx.x;

  // global index
  //const unsigned int sample_nr = bid*num_threads + tid;
 
  /* TG adding histogramming array GPU_HIS_DATA at beginning of gpu_odata */
  GPU_HIS_DATA *hisdata = (GPU_HIS_DATA*) gpu_odata;
  GPU_AUX_DATA *auxdata = (GPU_AUX_DATA*) (hisdata+1);
  /* end TG */
  //GPU_AUX_DATA *auxdata = (GPU_AUX_DATA*) gpu_odata;

  int sum;
  int idet;

  for (idet=0; idet<N_DETECTORS; idet++)
    {
      int i = tid + bid*num_threads; 
      while ( i < DEV_N_samples )
	{
	  ADC_TYPE adc = gpu_idata[idet*DEV_N_samples + i];
	  //adc = adc & 0x8000 ? (int) (adc&0x7FFF)-0x8000 : adc;
	  atomicAdd( &(data->ADC[idet][adc]), 1);
	  sum += adc;
	  i += blockDim.x * gridDim.x;
	}
    }

}
#endif

__global__
void kernel_print_map(ADC_TYPE *gpu_idata, ADC_TYPE* gpu_odata)
{
  /* TG adding histogramming array GPU_HIS_DATA at beginning of gpu_odata */
  GPU_HIS_DATA *hisdata = (GPU_HIS_DATA*) gpu_odata;
  GPU_AUX_DATA *auxdata = (GPU_AUX_DATA*) (hisdata+1);
  /* end TG */

  // access thread id
  const unsigned int tidx = threadIdx.x;
  const unsigned int tidy = threadIdx.y;


  //cuPrintf doesn't work on the Fermilab system
  //cuPrintf("kernel_print_map: thread.x %d, thread.y %d, module %d, channel %d\n", 
	// tidx, tidy, SegXYtoRiderModu[tidx][tidy], SegXYtoRiderChan[tidx][tidy]);
  //cuPrintf("kernel_print_map: structure nrmh %d, nrmt %d, nrch %d, nrct %d\n", 
  //	   RiderParams.nrmhwords, RiderParams.nrmtwords, RiderParams.nrchwords, RiderParams.nrctwords);
  //cuPrintf("kernel_print_map: DEV_first_sample_index %d dev_thres %d dev_decimation %d nrmh %d, nrmt %d, nrch %d, nrct %d\n", 
  //	   DEV_first_sample_index[tidx+9*tidy], DEV_threshold, DEV_decimation, NRMH_WORDS, NRMT_WORDS, NRCH_WORDS, NRCT_WORDS );
  //cuPrintf("kernel_print_map: xsegment %d, ysegment %d, idet %d,  pedestal %f\n", 
  //	   tidx, tidy, tidx+9*tidy, auxdata->pedestal[tidx+9*tidy]);

}

/** 
 * Make a fill-by-fill sum of waveforms in each detector / segment
 * 
 * @param gpu_idata 
 * @param gpu_odata 
 */
__global__
void kernel_wf_be64tole16(ADC_TYPE *gpu_idata, ADC_TYPE* gpu_odata)
{
  // access thread id
  const unsigned int tid = threadIdx.x;
  // access number of threads in this block
  const unsigned int num_threads = blockDim.x;
  // access block id
  const unsigned int bid = blockIdx.x;

  /* TG adding histogramming array GPU_HIS_DATA at beginning of gpu_odata */
  GPU_HIS_DATA *hisdata = (GPU_HIS_DATA*) gpu_odata;
  GPU_AUX_DATA *auxdata = (GPU_AUX_DATA*) (hisdata+1);
  /* end TG */
  //GPU_AUX_DATA *auxdata = (GPU_AUX_DATA*) gpu_odata;
  GPU_OUT_DATA *outdata = (GPU_OUT_DATA*) (auxdata+1);

  int sampletimesdetector_nr = 4 * ( tid + bid*num_threads ); 
  if ( sampletimesdetector_nr < DEV_N_samples*N_DETECTORS ) { 
    
    // re-order the bytes within 2-byte words
    unsigned int iByteReorder, lobyte, hibyte, four2bytewords[4];
    for (int iByteReorder = 0; iByteReorder  < 4; iByteReorder++ ){
      hibyte = (gpu_idata[sampletimesdetector_nr + iByteReorder] & 0xff00) >> 8;
      lobyte = (gpu_idata[sampletimesdetector_nr + iByteReorder] & 0xff);
      
      four2bytewords[iByteReorder] = lobyte << 8 | hibyte;
      
      //wg added mask for rider data, 7/6/15			
      four2bytewords[iByteReorder] = four2bytewords[iByteReorder] & 0x0fff;
    }
    // re-order the 2-byte words within 8-byte words
    for (int iByteReorder = 0; iByteReorder  < 4; iByteReorder++ ){
      gpu_idata[sampletimesdetector_nr + (3 - iByteReorder) ] = four2bytewords[iByteReorder];
    }
  }
}

/** 
 * Make a fill-by-fill sum of waveforms in each detector / segment
 * 
 * @param gpu_idata 
 * @param gpu_odata 
 */
__global__
void kernel_wf_fillsum(ADC_TYPE *gpu_idata, ADC_TYPE* gpu_odata)
{
  // access thread id
  const unsigned int tid = threadIdx.x;
  // access number of threads in this block
  const unsigned int num_threads = blockDim.x;
  // access block id
  const unsigned int bid = blockIdx.x;

  /* TG adding histogramming array GPU_HIS_DATA at beginning of gpu_odata */
  GPU_HIS_DATA *hisdata = (GPU_HIS_DATA*) gpu_odata;
  GPU_AUX_DATA *auxdata = (GPU_AUX_DATA*) (hisdata+1);
  /* end TG */
  //GPU_AUX_DATA *auxdata = (GPU_AUX_DATA*) gpu_odata;
  GPU_OUT_DATA *outdata = (GPU_OUT_DATA*) (auxdata+1);

  int sampletimesdetector_nr = tid + bid*num_threads; 
  if ( sampletimesdetector_nr < DEV_N_samples*N_DETECTORS ){

    int sample_nr = sampletimesdetector_nr%DEV_N_samples;
    int idet = sampletimesdetector_nr/DEV_N_samples;

    /*
    int irht = 0;
#ifdef USE_RIDER_FORMAT
    irht = NRMH_WORDS*(idet/N_RIDERCHANS+1)
      + NRMT_WORDS*(idet/N_RIDERCHANS)
      + NRCH_WORDS*(idet+1)
      + NRCT_WORDS*idet;
#endif

    hisdata->wf_hist[sampletimesdetector_nr] += gpu_idata[idet*DEV_N_samples + irht + sample_nr]; // not using map
    */

    ADC_TYPE adc = gpu_idata[DEV_first_sample_index[idet] + sample_nr];
    //adc = adc & 0x8000 ? (int) (adc&0x7FFF)-0x8000 : adc;
    //hisdata->wf_hist[sampletimesdetector_nr] += gpu_idata[DEV_first_sample_index[idet] + sample_nr]; // using map
    hisdata->wf_hist[sampletimesdetector_nr] += adc; // using map

  }
}

/** 
 * Make a sum of waveforms
 * 
 * @param gpu_idata 
 * @param gpu_odata 
 */
__global__
void kernel_wf_sum(ADC_TYPE *gpu_idata, ADC_TYPE* gpu_odata)//, int adc_sum_threshold)
{
  // access thread id
  const unsigned int tid = threadIdx.x;
  // access number of threads in this block
  const unsigned int num_threads = blockDim.x;
  // access block id
  const unsigned int bid = blockIdx.x;
  // global index
  //const unsigned int sample_nr = bid*num_threads + tid;

  /* TG adding histogramming array GPU_HIS_DATA at beginning of gpu_odata */
  GPU_HIS_DATA *hisdata = (GPU_HIS_DATA*) gpu_odata;
  GPU_AUX_DATA *auxdata = (GPU_AUX_DATA*) (hisdata+1);
  /* end TG */
  //GPU_AUX_DATA *auxdata = (GPU_AUX_DATA*) gpu_odata;
  GPU_OUT_DATA *outdata = (GPU_OUT_DATA*) (auxdata+1);

  int sample_nr = tid + bid*num_threads; 

  while ( sample_nr < DEV_N_samples )
    {
      double adc_sum = 0;
      unsigned int idet;
      for (idet=0; idet<N_DETECTORS; idet++)
	{

	  //adc_sum += A_calib[idet]*(auxdata->pedestal[idet]-gpu_idata[idet*DEV_N_samples + sample_nr]);
	  /* 
	  int irht = 0;
#ifdef USE_RIDER_FORMAT
	  irht = NRMH_WORDS*(idet/N_RIDERCHANS+1)
    	    + NRMT_WORDS*(idet/N_RIDERCHANS)
    	    + NRCH_WORDS*(idet+1)
    	    + NRCT_WORDS*idet;
#endif
	  adc_sum += A_calib[idet]*(auxdata->pedestal[idet]-gpu_idata[idet*DEV_N_samples + irht + sample_nr]);
	  */

	  ADC_TYPE adc = gpu_idata[DEV_first_sample_index[idet] + sample_nr];
	  //adc = adc & 0x8000 ? (int) (adc&0x7FFF)-0x8000 : adc;
	  adc_sum += A_calib[idet]*(adc - auxdata->pedestal[idet]);
	}      
      //cuPrintf("kernel_wf_sum:  idet %d, sample %d, pedestal %f, seg adc %d, adc_sum %f\n", 
      //	       1, sample_nr, auxdata->pedestal[0], gpu_idata[DEV_first_sample_index[0] + sample_nr], adc_sum );
      auxdata->wf_sum[sample_nr] = adc_sum;

      //auxdata->wf_sum[sample_nr] = 0;
      //for (idet=0; idet<N_DETECTORS; idet++)
      //auxdata->wf_sum[sample_nr] += auxdata->pedestal[idet]*100;
      
      // record island pattern
      //const int adc_sum_threshold = 100;
      //const int adc_sum_threshold = 200;
 
      //      if(adc_sum>600 && sample_nr>40000) atomicAdd(&(outdata->CTAG),1); //adc>2 GeV && t>50us
      //outdata->CTAG = 4321;

      if ( DEV_island_option==1 && DEV_threshold_sign && adc_sum > DEV_threshold )
	{
	  //  cuPrintf("kernel_wf_sum: trigger! adc_sum %f, sample %d\n", adc_sum, sample_nr );
	  auxdata->island_pattern[sample_nr] = 1;
	}
      if ( DEV_island_option==1 && !DEV_threshold_sign && adc_sum < DEV_threshold )
	{
	  //  cuPrintf("kernel_wf_sum: trigger! adc_sum %f, sample %d\n", adc_sum, sample_nr );
	  auxdata->island_pattern[sample_nr] = 1;
	}
      
      // period based trigger for tests with amc13 fake data, 29 Sept 2014, TG
      const int trigger_period = 50000; // period chosen to match hit rate in the g-2 experiment
      if(DEV_island_option==0 && sample_nr%trigger_period == 0 )
	{      
	  auxdata->island_pattern[sample_nr] = 1;
	}
      
      sample_nr += blockDim.x * gridDim.x;
    }

}

/** 
 * Find the triggers in waveforms
 * 
 * @param gpu_idata 
 * @param gpu_odata 
 */
__global__
void kernel_wf_trigger(ADC_TYPE *gpu_idata, ADC_TYPE* gpu_odata)
{
  // input / aux / output data arrays
  GPU_HIS_DATA *hisdata = (GPU_HIS_DATA*) gpu_odata;
  GPU_AUX_DATA *auxdata = (GPU_AUX_DATA*) (hisdata+1);
  GPU_OUT_DATA *outdata = (GPU_OUT_DATA*) (auxdata+1);

  // access thread id, block id, .. to define the sample index
  const unsigned int tid = threadIdx.x;
  const unsigned int bid = blockIdx.x;
  const unsigned int num_threads = blockDim.x;

  int sample_nr = tid + bid*num_threads;

  while ( sample_nr < DEV_N_samples )
    {
      
      unsigned int idet;
      
      // leading edge threshold on individual segments
      if ( DEV_island_option == 2 ){
	double adc_cal = 0;
	for (idet=0; idet<N_DETECTORS; idet++)
	  {
	    ADC_TYPE adc = gpu_idata[DEV_first_sample_index[idet] + sample_nr];
	    //adc = adc & 0x8000 ? (int) (adc&0x7FFF)-0x8000 : adc;
	    adc_cal = A_calib[idet]*(adc - auxdata->pedestal[idet]);

	    if ( DEV_threshold_sign && adc_cal > DEV_threshold ){
	      auxdata->island_pattern[sample_nr] = 1;
	      break;
	    }
	    if ( !DEV_threshold_sign && adc_cal < DEV_threshold ){
	      auxdata->island_pattern[sample_nr] = 1;
	      break;
	    }
	  }      
      }	

      // pulseshape weighted threshold on individual segments
      if ( DEV_island_option == 3 ){

	const int Nwgt = 7;
	double wgt[Nwgt] = {0.0625, 0.1250, 0.1875, 0.2500, 0.1875, 0.1250, 0.0625};
	int wgtlo = -3, wgthi = 3; 	  
	
	if ( (sample_nr > -wgtlo) && (sample_nr < (DEV_N_samples - wgthi)) ){  // bookend

	  for (idet=0; idet<N_DETECTORS; idet++)
	    {
	      int firstsample = DEV_first_sample_index[idet];
	      double calconst = A_calib[idet];
	      double pedval = auxdata->pedestal[idet];
	      
	      double adcwgt = 0.0;
              int iwgt = 0;
	      for (iwgt = wgtlo; iwgt <= wgthi; iwgt++){
	        ADC_TYPE adc =  gpu_idata[firstsample + sample_nr - iwgt];
	        //adc = adc & 0x8000 ? (int) (adc&0x7FFF)-0x8000 : adc;
		adcwgt += wgt[iwgt - wgtlo] * calconst * (adc - pedval);
	      }

	      if ( DEV_threshold_sign && adcwgt > DEV_threshold ){
		auxdata->island_pattern[sample_nr] = 1;
		break;
	      }
	      if ( !DEV_threshold_sign && adcwgt < DEV_threshold ){
		auxdata->island_pattern[sample_nr] = 1;
		break;
	      }
	    }
	}
      
      }	 
      sample_nr += blockDim.x * gridDim.x;
    }

}

__global__
void kernel_extend_islands(ADC_TYPE *gpu_idata, ADC_TYPE* gpu_odata)
{
  // access thread id
  const unsigned int tid = threadIdx.x;
  // access number of threads in this block
  const unsigned int num_threads = blockDim.x;
  // access block id
  const unsigned int bid = blockIdx.x;
  // global index
  //const unsigned int sample_nr = bid*num_threads + tid;

  /* TG adding histogramming array GPU_HIS_DATA at beginning of gpu_odata */
  GPU_HIS_DATA *hisdata = (GPU_HIS_DATA*) gpu_odata;
  GPU_AUX_DATA *auxdata = (GPU_AUX_DATA*) (hisdata+1);
  /* end TG */
  //GPU_AUX_DATA *auxdata = (GPU_AUX_DATA*) gpu_odata;
  GPU_OUT_DATA *outdata = (GPU_OUT_DATA*) (auxdata+1);

  //const int n_presamples = 10;
  //const int n_postsamples = 15;

  int sample_nr = tid + bid*num_threads; 
  while ( sample_nr < DEV_N_samples )
    {
      
      int is_BOI = 0; // beginning of an island
      int is_EOI = 0; // end of an island

      // check the BOI and EOI conditions
      if ( auxdata->island_pattern[sample_nr] > 0 )
	{
	  // check BOI condition
	  if ( sample_nr == 0 ) 
	    is_BOI=1;
	  else
	    if ( auxdata->island_pattern[sample_nr-1] == 0 )
	      is_BOI=1;
	  
	   // check EOI condition
	  if ( sample_nr == (DEV_N_samples-1) )
	    is_EOI=1;
	  else
	    if ( auxdata->island_pattern[sample_nr+1] == 0 )
	      is_EOI=1;
	}
	
      if ( is_BOI )
	{
	  // This is a beginning of an island
	  // extend the island for N_presamples
	  int i1 = sample_nr - DEV_N_presamples;
	  if ( i1 < 0 ) i1 = 0;
	  int k;
	  for (k=i1; k<sample_nr; k++)
	    {
	      atomicAdd( &(auxdata->island_pattern[k]), 1);		  
	    }
	}

      // check the "End Of Island" condition
      if ( is_EOI )
	{
	  // This is an end of an island
	  // extend the island for N_postsamples
	  int i2 = sample_nr + DEV_N_postsamples;
	  if ( i2 >= DEV_N_samples ) i2 = DEV_N_samples-1;
	  int k;
	  for (k=i2; k>sample_nr; k--)
	    {
	      atomicAdd( &(auxdata->island_pattern[k]), 1);		  
	    }
	}

      sample_nr += blockDim.x * gridDim.x;

    }

}

__global__
void kernel_find_islands(ADC_TYPE *gpu_idata, ADC_TYPE* gpu_odata)
{
  // access thread id
  const unsigned int tid = threadIdx.x;
  // access number of threads in this block
  const unsigned int num_threads = blockDim.x;
  // access block id
  const unsigned int bid = blockIdx.x;
  // global index
  //const unsigned int sample_nr = bid*num_threads + tid;

  /* TG adding histogramming array GPU_HIS_DATA at beginning of gpu_odata */
  GPU_HIS_DATA *hisdata = (GPU_HIS_DATA*) gpu_odata;
  GPU_AUX_DATA *auxdata = (GPU_AUX_DATA*) (hisdata+1);
  /* end TG */
  //GPU_AUX_DATA *auxdata = (GPU_AUX_DATA*) gpu_odata;
  GPU_OUT_DATA *outdata = (GPU_OUT_DATA*) (auxdata+1);


  //find new islands (some of the old island could have merged 
  
  int sample_nr = tid + bid*num_threads; 
  while ( sample_nr < DEV_N_samples )
    {

      int is_BOI = 0; // beginning of an island

      // check the BOI and EOI conditions
      if ( auxdata->island_pattern[sample_nr] > 0 )
	{
	  // check BOI condition
	  if ( sample_nr == 0 ) 
	    is_BOI=1;
	  else
	    if ( auxdata->island_pattern[sample_nr-1] == 0 )
	      is_BOI=1;
	}


      if ( is_BOI )
	{
	  // This is a beginning of an island
	  
	  // island number
	  int island_nr = atomicAdd( &(outdata->n_islands), 1); 
	  auxdata->island_info[island_nr].time = sample_nr;
	  
	  // determine the length of the island
	  int i;
	  int island_nr_aux = island_nr + 1;
	  for (i=sample_nr; i<DEV_N_samples; i++)
	    {
	      if ( auxdata->island_pattern[i] == 0 )
		{
		  break;
		}
	      else
		{
		  auxdata->island_pattern[i] = island_nr_aux;
		}
	    }
	  int island_len = i - sample_nr;
	  // record the length into the first bin
	  //data->island_pattern[sample_nr] = -island_len;
	  auxdata->island_info[island_nr].length = island_len;
	  int offset = atomicAdd( &(outdata->island_offset), 4 + N_DETECTORS*island_len);
	  auxdata->island_info[island_nr].offset = offset;
	  
	  outdata->islands[offset+0] = (int16_t) sample_nr;
	  outdata->islands[offset+1] = (int16_t) ( sample_nr >> 16 );
	  outdata->islands[offset+2] = (int16_t) island_len;
	  outdata->islands[offset+3] = (int16_t) ( island_len >> 16 );

	  //outdata->islands[offset]   = (0x0000FFFF && sample_nr);
	  //outdata->islands[offset+2] = island_len; // +2 as sample_nr and island_length are 32-bit words in 16-bit array
	  
	}

      sample_nr += blockDim.x * gridDim.x; 
    }

}


__global__
void kernel_make_islands(ADC_TYPE *gpu_idata, ADC_TYPE* gpu_odata)
{
  // access thread id
  const unsigned int tid = threadIdx.x;
  // access number of threads in this block
  const unsigned int num_threads = blockDim.x;
  // access block id
  const unsigned int bid = blockIdx.x;
  // global index
  //const unsigned int sample_nr = bid*num_threads + tid;

  /* TG adding histogramming array GPU_HIS_DATA at beginning of gpu_odata */
  GPU_HIS_DATA *hisdata = (GPU_HIS_DATA*) gpu_odata;
  GPU_AUX_DATA *auxdata = (GPU_AUX_DATA*) (hisdata+1);
  /* end TG */
  //GPU_AUX_DATA *auxdata = (GPU_AUX_DATA*) gpu_odata;
  GPU_OUT_DATA *outdata = (GPU_OUT_DATA*) (auxdata+1);


  //find new islands (some of the old island could have merged 
  
  int sample_nr = tid + bid*num_threads; 
  while ( sample_nr < DEV_N_samples )
    {
      int island_nr = auxdata->island_pattern[sample_nr];
      if ( island_nr > 0 )
	{
	  island_nr--;
	  int island_offset    = auxdata->island_info[island_nr].offset + 4; // +4 to to skip 32-bit time and length words in 16-bit array 
	  int island_sample_nr = sample_nr - auxdata->island_info[island_nr].time;
	  int island_length    = auxdata->island_info[island_nr].length;
	  int idet;
	  for (idet=0; idet<N_DETECTORS; idet++)
	    {
	      int i = island_offset + idet*island_length + island_sample_nr;

	      /*
#ifdef USE_RIDER_FORMAT
	      int irht = 0;
              irht = NRMH_WORDS*(idet/N_RIDERCHANS+1)
    		+ NRMT_WORDS*(idet/N_RIDERCHANS)
    		+ NRCH_WORDS*(idet+1)
    		+ NRCT_WORDS*idet;
#endif
	      outdata->islands[i] = gpu_idata[idet*DEV_N_samples + irht + sample_nr];
	      */
	      ADC_TYPE adc=gpu_idata[DEV_first_sample_index[idet] + sample_nr];
	      //adc = adc & 0x8000 ? (int) (adc&0x7FFF)-0x8000 : adc;
	      outdata->islands[i] = adc;
	    }
	}
      sample_nr += blockDim.x * gridDim.x; 
    }

#if 0
  // record samples above a certain threshold
  if ( cal_data->wf_sum.adc[sample_nr] < 8950 )
    {
      //cal_data->wf_sum_thr.adc[sample_nr] = cal_data->wf_sum.adc[sample_nr];
    }
#endif

#if 0

  sample_nr = tid + bid*num_threads; 
  while ( sample_nr < DEV_N_samples )
    {
      auxdata->island_pattern[sample_nr] = sample_nr;
      sample_nr += blockDim.x * gridDim.x; 
    }
#endif


  /*
  if ( tid == 1 && bid == 1 )
    {
      auxdata->island_pattern[0] = num_threads;
      auxdata->island_pattern[1] = blockDim.x;
      auxdata->island_pattern[2] = gridDim.x;
      auxdata->island_pattern[3] = sizeof(int);
    }
  */
}


/** 
 * Evaluate pedestals
 * Use last 500 samples for averaging
 * 
 * @param gpu_idata 
 * @param gpu_odata 
 */
__global__
void kernel_make_pedestals(ADC_TYPE *gpu_idata, ADC_TYPE* gpu_odata)
{

  // access thread id
  const unsigned int tid = threadIdx.x;
  // access number of threads in this block
  const unsigned int num_threads = blockDim.x;
  // access block id
  const unsigned int bid = blockIdx.x;

  // global index
 
  /* TG adding histogramming array GPU_HIS_DATA at beginning of gpu_odata */
  GPU_HIS_DATA *hisdata = (GPU_HIS_DATA*) gpu_odata;
  GPU_AUX_DATA *auxdata = (GPU_AUX_DATA*) (hisdata+1);
  /* end TG */
  //GPU_AUX_DATA *auxdata = (GPU_AUX_DATA*) gpu_odata;

  int idet = bid;                // detector number 
  int offset = idet*DEV_N_samples;   // offset in the input data array

  if (DEV_pedestal_option == 1) {

    const int nsamples = 500;     // number of samples for averaging
    double adc_mean = 0; // mean value of adc samples
    int i;
    for (i=0; i<nsamples; i++)
      {
	
	/*
	  int irht = 0;
	  #ifdef USE_RIDER_FORMAT
	  irht = NRMH_WORDS*(idet/N_RIDERCHANS+1)
	  + NRMT_WORDS*(idet/N_RIDERCHANS)
	  + NRCH_WORDS*(idet+1)
	  + NRCT_WORDS*idet;
	  #endif 
	  adc_mean += gpu_idata[DEV_N_samples-i+offset+irht]; // use last nsamples samples
	*/
	ADC_TYPE adc = gpu_idata[DEV_first_sample_index[idet]+DEV_N_samples-i-1];
	//adc = adc & 0x8000 ? (int) (adc&0x7FFF)-0x8000 : adc;
	adc_mean += adc; // use last nsamples samples
      }
    adc_mean /= nsamples;
    auxdata->pedestal[idet] = adc_mean;
    
  }

  if (DEV_pedestal_option == 0) {

    auxdata->pedestal[idet] = DEV_global_pedestal;

  }
}

__global__
void kernel_calc_ctag( void* gpu_odata)
{
  // access thread id
  const unsigned int tid = threadIdx.x;
  // access number of threads in this block
  const unsigned int num_threads = blockDim.x;
  // access block id
  const unsigned int bid = blockIdx.x;
  // global index
  const unsigned int sample_nr = bid*num_threads + tid;

  /* TG adding histogramming array GPU_HIS_DATA at beginning of gpu_odata */
  GPU_HIS_DATA *hisdata = (GPU_HIS_DATA*) gpu_odata;
  GPU_AUX_DATA *auxdata = (GPU_AUX_DATA*) (hisdata+1);
  /* end TG */
  //GPU_AUX_DATA *auxdata = (GPU_AUX_DATA*) gpu_odata;
  GPU_OUT_DATA *outdata = (GPU_OUT_DATA*) (auxdata+1);

  //int found_ctag = 0;
  //int i;
  //for (i=0; i<sample_nr; i++)
  //{
  int threshold_low = 600;
  int threshold_high = 1000;
  if(sample_nr>40000 && sample_nr<N_SAMPLES_MAX && auxdata->wf_sum[sample_nr]>threshold_high)
    if( auxdata->wf_sum[sample_nr]>auxdata->wf_sum[sample_nr+1] && auxdata->wf_sum[sample_nr]>auxdata->wf_sum[sample_nr-1])//ONLY COUNT THE PEAK
      atomicAdd(&(outdata->CTAG),1); //adc>2 GeV && t>50us
  
}
  

__global__
void kernel_decimate_sum( void* gpu_odata)
{
  // access thread id
  const unsigned int tid = threadIdx.x;
  // access number of threads in this block
  const unsigned int num_threads = blockDim.x;
  // access block id
  const unsigned int bid = blockIdx.x;
  // global index
  //const unsigned int sample_nr = bid*num_threads + tid;

  /* TG adding histogramming array GPU_HIS_DATA at beginning of gpu_odata */
  GPU_HIS_DATA *hisdata = (GPU_HIS_DATA*) gpu_odata;
  GPU_AUX_DATA *auxdata = (GPU_AUX_DATA*) (hisdata+1);
  /* end TG */
  //GPU_AUX_DATA *auxdata = (GPU_AUX_DATA*) gpu_odata;
  GPU_OUT_DATA *outdata = (GPU_OUT_DATA*) (auxdata+1);

  int sample_nr_1 = (tid + bid*num_threads)*DEV_decimation;
  int sample_nr_2 = sample_nr_1 +  DEV_decimation;
  int i;
  double adc_sum = 0;
  for (i=sample_nr_1; i<sample_nr_2; i++)
    {
      adc_sum += auxdata->wf_sum[i];
      //break; // makes the sum just first sample of DEV_decimation samples
    }
  
  // append the sum to the end of the output data
  //int *data = (int*)(outdata) + (2 + outdata->island_offset*sizeof(int));
  int16_t *data = (int16_t*)(outdata) + (6 + outdata->island_offset); // now 6 16-bit header words for data size, island number, CTAG
  data[ sample_nr_1 / DEV_decimation ] = (int16_t) ( adc_sum / DEV_decimation );

  // test decimate routine by writing out first sample of added samples  sample_nr_1
  //data[sample_nr_1/DEV_decimation] = (int16_t) sample_nr_1;


}

void cuda_g2_bor_kernel(){

  hipError_t cudaCopyStatus;

  printf("cuda_g2_bor_kernel()\n");

  HOST_N_samples = amc13_settings_odb.gpu_waveform_length;
  HOST_N_presamples = amc13_settings_odb.gpu_island_presamples;
  HOST_N_postsamples = amc13_settings_odb.gpu_island_postsamples;
  HOST_decimation = amc13_settings_odb.calosum_decimation_factor;
  HOST_island_option = amc13_settings_odb.island_option;
  HOST_threshold = amc13_settings_odb.T_threshold;
  HOST_threshold_sign = amc13_settings_odb.T_threshold_sign;
  HOST_pedestal_option = amc13_settings_odb.pedestal_option;
  HOST_global_pedestal = amc13_settings_odb.global_pedestal;

  amc13_ODB_get();
   
  // calculate the array index of first sample of each calo segment
  int index = 0;
  int ix, iy, im, ic;
  for(im=0;im<12;im++){
    if (amc13_rider_odb[im].board.rider_enabled) {
      
#ifdef USE_RIDER_FORMAT
      index += NRMH_WORDS;
#endif

      for(ic=0;ic<5;ic++){
	if (amc13_rider_odb[im].channel[ic].enabled) {
	  
#ifdef USE_RIDER_FORMAT
	  index += NRCH_WORDS;
#endif
	  
	  ix = rider_map_to_calo_odb[im][ic].calo_segx;
	  iy = rider_map_to_calo_odb[im][ic].calo_segy;

	  if (ix >= 1 && ix <= 9 && iy >= 1 && iy <= 6) {
	    HOST_first_sample_index[ (ix-1) + (iy-1)*N_SEGMENTS_X ] = index;
	    printf("calo segment x,y %i, %i  first_sample_index %i\n", ix, iy, HOST_first_sample_index[ (ix-1) + (iy-1)*N_SEGMENTS_X ] );
	  }
	  
          // temporarily set all channels to gpu_waveform_length
          // until more details of odb structure for rider boards
          // and channels is available
	  // index += amc13_rider_odb[im].board.sample_length;
	  index += amc13_settings_odb.gpu_waveform_length;  

#ifdef USE_RIDER_FORMAT
	  index += NRCT_WORDS;
#endif
	}
      }
      
#ifdef USE_RIDER_FORMAT
      index += NRMT_WORDS;
#endif
    }
  }
    
  // copy gpu analysis paramters to device

  cudaCopyStatus = hipMemcpyToSymbol(HIP_SYMBOL( DEV_N_samples), &HOST_N_samples, sizeof(HOST_N_samples), 0, hipMemcpyHostToDevice);
  if ( cudaCopyStatus != hipSuccess )
    {
      printf("hipMemcpyToSymbol of N_samples FAIL, bytes %d \n", sizeof(HOST_N_samples));
    }
  printf("hipMemcpyToSymbol of number of samples %i, status %i \n", HOST_N_samples, (int)cudaCopyStatus );

  cudaCopyStatus = hipMemcpyToSymbol(HIP_SYMBOL( DEV_N_presamples), &HOST_N_presamples, sizeof(HOST_N_presamples), 0, hipMemcpyHostToDevice);
  if ( cudaCopyStatus != hipSuccess )
    {
      printf("hipMemcpyToSymbol of N_presamples FAIL, bytes %d \n", sizeof(HOST_N_presamples));
    }
  printf("hipMemcpyToSymbol of number of island pre-samples %i, status %i \n", HOST_N_presamples, (int)cudaCopyStatus );

  cudaCopyStatus = hipMemcpyToSymbol(HIP_SYMBOL( DEV_N_postsamples), &HOST_N_postsamples, sizeof(HOST_N_postsamples), 0, hipMemcpyHostToDevice);
  if ( cudaCopyStatus != hipSuccess )
    {
      printf("hipMemcpyToSymbol of N_postsamples FAIL, bytes %d \n", sizeof(HOST_N_postsamples));
    }
  printf("hipMemcpyToSymbol of number of island post-samples %i, status %i \n", HOST_N_postsamples, (int)cudaCopyStatus );

  cudaCopyStatus = hipMemcpyToSymbol(HIP_SYMBOL( DEV_decimation), &HOST_decimation, sizeof(HOST_decimation), 0, hipMemcpyHostToDevice);
  if ( cudaCopyStatus != hipSuccess )
    {
      printf("hipMemcpyToSymbol of decimation FAIL, bytes %d \n", sizeof(HOST_decimation));
    }
  printf("hipMemcpyToSymbol of decimation factor %i, status %i \n", HOST_decimation, (int)cudaCopyStatus );

  cudaCopyStatus = hipMemcpyToSymbol(HIP_SYMBOL( DEV_first_sample_index), HOST_first_sample_index, N_DETECTORS*sizeof(int), 0, hipMemcpyHostToDevice);
  if ( cudaCopyStatus != hipSuccess )
    {
      printf("hipMemcpyToSymbol of first sample indices FAIL, bytes %d \n", sizeof(HOST_first_sample_index));
    }
  printf("hipMemcpyToSymbol of first sample indices %i, status %i \n", HOST_first_sample_index[0], (int)cudaCopyStatus );

 cudaCopyStatus = hipMemcpyToSymbol(HIP_SYMBOL( DEV_island_option), &HOST_island_option, sizeof(HOST_island_option), 0, hipMemcpyHostToDevice);
   if ( cudaCopyStatus != hipSuccess )
     {
        printf("hipMemcpyToSymbol of island_options FAIL, bytes %d \n", sizeof(HOST_island_option));
    }	
  printf("hipMemcpyToSymbol of island_option %i, status %i \n", HOST_island_option, (int)cudaCopyStatus );

  cudaCopyStatus = hipMemcpyToSymbol(HIP_SYMBOL( DEV_threshold), &HOST_threshold, sizeof(HOST_threshold), 0, hipMemcpyHostToDevice);
   if ( cudaCopyStatus != hipSuccess )
     {
        printf("hipMemcpyToSymbol of thresholds FAIL, bytes %d \n", sizeof(HOST_threshold));
    }	
  printf("hipMemcpyToSymbol of threshold %i, status %i \n", HOST_threshold, (int)cudaCopyStatus );

  cudaCopyStatus = hipMemcpyToSymbol(HIP_SYMBOL( DEV_threshold_sign), &HOST_threshold_sign, sizeof(HOST_threshold_sign), 0, hipMemcpyHostToDevice);
   if ( cudaCopyStatus != hipSuccess )
     {
        printf("hipMemcpyToSymbol of threshold sign FAIL, bytes %d \n", sizeof(HOST_threshold_sign));
    }	
  printf("hipMemcpyToSymbol of threshold sign%i, status %i \n", HOST_threshold_sign, (int)cudaCopyStatus );

 cudaCopyStatus = hipMemcpyToSymbol(HIP_SYMBOL( DEV_pedestal_option), &HOST_pedestal_option, sizeof(HOST_pedestal_option), 0, hipMemcpyHostToDevice);
   if ( cudaCopyStatus != hipSuccess )
     {
        printf("hipMemcpyToSymbol of pedestal_options FAIL, bytes %d \n", sizeof(HOST_pedestal_option));
    }	
  printf("hipMemcpyToSymbol of pedestal_option %i, status %i \n", HOST_pedestal_option, (int)cudaCopyStatus );

 cudaCopyStatus = hipMemcpyToSymbol(HIP_SYMBOL( DEV_global_pedestal), &HOST_global_pedestal, sizeof(HOST_global_pedestal), 0, hipMemcpyHostToDevice);
   if ( cudaCopyStatus != hipSuccess )
     {
        printf("hipMemcpyToSymbol of global_pedestals FAIL, bytes %d \n", sizeof(HOST_global_pedestal));
    }	
  printf("hipMemcpyToSymbol of global_pedestal %i, status %i \n", HOST_global_pedestal, (int)cudaCopyStatus );


  return;
}

/*
 * gpu_idata input data of coninuous samples from gpu_thread to GPU
 * cpu_odata output data of T-method, Q-method to gpu_thread from GPU
 */

//void cuda_g2_run_kernel( unsigned char *gpu_idata, unsigned char *gpu_odata )
void cuda_g2_run_kernel( unsigned char *gpu_idata, unsigned char *gpu_odata, 
			 int16_t *cpu_odata )
{
  hipError_t cudaCopyStatus;

  const int n_threads_per_block = 1024; //wg changed from 256, 4/2/14
  //const int n_threads_per_block = 512; // tg test for c1060 + k20 two readout per FE 2 March 2015

  // get GPU waveform length from odb parameters
  HOST_N_samples = amc13_settings_odb.gpu_waveform_length;

  int n_blocks = HOST_N_samples / n_threads_per_block;
  //int n_blocks = 480; //changed to above on 4/2/14, wg
  if ( n_blocks < 1 ) n_blocks = 1;
  dim3  grid( n_blocks, 1, 1);
  dim3  threads( n_threads_per_block, 1, 1);

  printf(" ::: start-of-kernel, size of  GPU_OBUF_SIZE 0x%08x, GPU_HIS_DATA 0x%08x, GPU_AUX_DATA 0x%08x, GPU_OUT_DATA 0x%08x \n", 
  	   GPU_OBUF_SIZE, sizeof(GPU_HIS_DATA), sizeof(GPU_AUX_DATA), sizeof(GPU_OUT_DATA) );

  // measure time
#define TIME_MEASURE_DEF
#ifdef TIME_MEASURE_DEF
  hipEvent_t start, stop;
  hipEvent_t start_all, stop_all;
  float elapsedTime;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipEventCreate(&start_all);
  hipEventCreate(&stop_all);
  hipEventRecord(start_all, 0);
#endif // TIME_MEASURE_DEF


  dbprintf(" ::: %d samples, %d blocks, %d threads/block, %d threads\n", 
	   HOST_N_samples, n_blocks, n_threads_per_block, n_blocks*n_threads_per_block);

  // reset the output memory
#ifdef TIME_MEASURE_DEF
  // start event
  hipEventRecord(start, 0);
#endif // TIME_MEASURE_DEF
  // avoid zeroing of fill-by-fill histogram
  GPU_HIS_DATA *hisdata = (GPU_HIS_DATA*) gpu_odata;
  GPU_AUX_DATA *auxdata = (GPU_AUX_DATA*) (hisdata+1);
  hipMemset( auxdata, 0, ( GPU_OBUF_SIZE - sizeof(GPU_HIS_DATA) ) );
#ifdef TIME_MEASURE_DEF
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsedTime, start, stop);
  printf(" ::: GPU_OBUF reset time %f ms (%i MB)\n",elapsedTime, GPU_OBUF_SIZE/1024/1024);
#endif // TIME_MEASURE_DEF

  // test of cudaMemCpyToSymbol

#ifdef TIME_MEASURE_DEF
  // start event
  hipEventRecord(start, 0);
#endif // TIME_MEASURE_DEF

  /* tests of copying structure to GPU - Sept, 2015

  // copy N_samples to GPU
  hipError_t cudaCopyStatus;
  cudaCopyStatus = hipMemcpyToSymbol(HIP_SYMBOL( DEV_N_samples), &HOST_N_samples, sizeof(HOST_N_samples), 0, hipMemcpyHostToDevice);
  if ( cudaCopyStatus != hipSuccess )
    {
      printf("hipMemcpyToSymbol of N_samples FAIL, bytes %d\n", sizeof(HOST_N_samples));
    }  

  HostRiderParams = (RIDERPARAMS*) malloc( sizeof(RIDERPARAMS) );
  HostRiderParams->nrmhwords = NRMH_WORDS;
  HostRiderParams->nrmtwords = NRMT_WORDS;
  HostRiderParams->nrchwords = NRCH_WORDS;
  HostRiderParams->nrctwords = NRCT_WORDS;
  hipMalloc( (void**) &RiderParams, sizeof(RIDERPARAMS));

  cudaCopyStatus = hipMemcpyToSymbol(HIP_SYMBOL( RiderParams), HostRiderParams, sizeof(RIDERPARAMS), 0, hipMemcpyHostToDevice);
  if ( cudaCopyStatus != hipSuccess )
    {
      printf("hipMemcpyToSymbol FAIL, bytes %d\n", sizeof(RIDERPARAMS));
    }

  cudaCopyStatus = hipMemcpyToSymbol(HIP_SYMBOL( SegXYtoRiderModu), HostSegXYtoRiderModu, N_DETECTORS*sizeof(int), 0, hipMemcpyHostToDevice);
  if ( cudaCopyStatus != hipSuccess )
    {
      printf("hipMemcpyToSymbol FAIL, bytes %d\n", N_DETECTORS*sizeof(int));
    }
  printf("hipMemcpyToSymbol of SegXYtoModu %i, status %i \n", HOST_first_sample_index[0], (int)cudaCopyStatus );

  cudaCopyStatus = hipMemcpyToSymbol(HIP_SYMBOL( SegXYtoRiderChan), HostSegXYtoRiderChan, N_DETECTORS*sizeof(int), 0, hipMemcpyHostToDevice);
  if ( cudaCopyStatus != hipSuccess )
    {
      printf("hipMemcpyToSymbol FAIL, bytes %d\n", N_DETECTORS*sizeof(int));
    }

#ifdef TIME_MEASURE_DEF
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsedTime, start, stop);
  printf(" ::: hipMemcpyToSymbol test  time %f ms\n",elapsedTime);
#endif // TIME_MEASURE_DEF

  cudaPrintfInit();
  dim3  grid_test( 1, 1, 1);
  dim3  threads_test( 9, 6, 1); 
  kernel_print_map<<< grid_test, threads_test >>>( (ADC_TYPE*)gpu_idata, (ADC_TYPE*)gpu_odata );
  cudaPrintfDisplay(stdout, true);
  cudaPrintfEnd();

  */

#if 0
  // re-order bytes in 16-bit ADC words for big-endian 64-bit AMC13 words
  // is necessary for real data, isnt necessary for emulator data as of 11 Sept 2015 / TG
#ifdef TIME_MEASURE_DEF
  // start event
  hipEventRecord(start, 0);
#endif // TIME_MEASURE_DEF
  
  int n_blocks_4 = ( HOST_N_samples * N_DETECTORS / n_threads_per_block ) / 4 + 1;
  dim3  grid_4( n_blocks_4, 1, 1);
  dim3  threads_4( n_threads_per_block, 1, 1);
  kernel_wf_be64tole16<<< grid_4, threads_4>>>( (ADC_TYPE*)gpu_idata, (ADC_TYPE*)gpu_odata );

#ifdef TIME_MEASURE_DEF
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsedTime, start, stop);
  printf(" ::: kernel_wf_be64tole16 time %f ms\n",elapsedTime);
#endif // TIME_MEASURE_DEF
#endif

#if 1
  // evaluate pedestals
#ifdef TIME_MEASURE_DEF
  // start event
  hipEventRecord(start, 0);
#endif // TIME_MEASURE_DEF
  dim3  grid_1   ( 54, 1, 1);  // 35 blocks (wg changed to 54 on 4/1/14)
  dim3  threads_1(  1, 1, 1);  // 1 thread
  kernel_make_pedestals<<< grid_1, threads_1>>>( (ADC_TYPE*)gpu_idata, (ADC_TYPE*)gpu_odata );
#ifdef TIME_MEASURE_DEF
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsedTime, start, stop);
  printf(" ::: kernel_make_pedestals time %f ms\n",elapsedTime);
#endif // TIME_MEASURE_DEF

  /*
  cudaPrintfInit();
  dim3  grid_test( 1, 1, 1);
  dim3  threads_test( 9, 6, 1); 
  kernel_print_map<<< grid_test, threads_test >>>( (ADC_TYPE*)gpu_idata, (ADC_TYPE*)gpu_odata );
  cudaPrintfDisplay(stdout, true);
  cudaPrintfEnd();
  */

#endif
  // make the distribution of ADC samples
  // slow. Don't use this.
  //kernel_wf_make_ADC<<< grid, threads>>>( (ADC_TYPE*)gpu_idata, (ADC_TYPE*)gpu_odata );


#if 1
  // Sum all waveforms
#ifdef TIME_MEASURE_DEF
  // start event
  hipEventRecord(start, 0);
#endif // TIME_MEASURE_DEF
  //int threshold = amc13_settings_odb.T_threshold;
  //cudaPrintfInit();
  kernel_wf_sum<<< grid, threads>>>( (ADC_TYPE*)gpu_idata, (ADC_TYPE*)gpu_odata);//  , threshold );
  //cudaPrintfDisplay(stdout, true);
  //cudaPrintfEnd();
#ifdef TIME_MEASURE_DEF
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsedTime, start, stop);
  printf(" ::: kernel_wf_sum time %f ms\n",elapsedTime);
#endif // TIME_MEASURE_DEF
#endif

#if 1
  // Sum all waveforms
#ifdef TIME_MEASURE_DEF
  // start event
  hipEventRecord(start, 0);
#endif // TIME_MEASURE_DEF
  //int threshold = amc13_settings_odb.T_threshold;
  //cudaPrintfInit();
  kernel_wf_trigger<<< grid, threads>>>( (ADC_TYPE*)gpu_idata, (ADC_TYPE*)gpu_odata);//  , threshold );
  //cudaPrintfDisplay(stdout, true);
  //cudaPrintfEnd();
#ifdef TIME_MEASURE_DEF
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsedTime, start, stop);
  printf(" ::: kernel_wf_trigger time %f ms\n",elapsedTime);
#endif // TIME_MEASURE_DEF
#endif

  //#if 0 // TG turned off for just Q-method tests, Sep 22, 2014
#if 1
#ifdef TIME_MEASURE_DEF
  // start event
  hipEventRecord(start, 0);
#endif // TIME_MEASURE_DEF
  // Extend islands by a predefined number of samples
  kernel_extend_islands<<< grid, threads>>>( (ADC_TYPE*)gpu_idata, (ADC_TYPE*)gpu_odata );
#ifdef TIME_MEASURE_DEF
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsedTime, start, stop);
  printf(" ::: kernel_extend_islands time %f ms\n",elapsedTime);
#endif // TIME_MEASURE_DEF
#endif
  

  //#if 0 // TG turned off for just Q-method tests, Sep 22, 2014
#if 1
  // Find islands
#ifdef TIME_MEASURE_DEF
  // start event
  hipEventRecord(start, 0);
#endif // TIME_MEASURE_DEF
  kernel_find_islands<<< grid, threads>>>( (ADC_TYPE*)gpu_idata, (ADC_TYPE*)gpu_odata );
#ifdef TIME_MEASURE_DEF
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsedTime, start, stop);
  printf(" ::: kernel_find_islands time %f ms\n",elapsedTime);
#endif // TIME_MEASURE_DEF

#endif

  //#if 0 // TG turned off for just Q-method tests, Sep 22, 2014
#if 1
  // Make islands
#ifdef TIME_MEASURE_DEF
  // start event
  hipEventRecord(start, 0);
#endif // TIME_MEASURE_DEF
  kernel_make_islands<<< grid, threads>>>( (ADC_TYPE*)gpu_idata, (ADC_TYPE*)gpu_odata );
#ifdef TIME_MEASURE_DEF
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsedTime, start, stop);
  printf(" ::: kernel_make_islands time %f ms\n",elapsedTime);
#endif // TIME_MEASURE_DEF
#endif

#if 1
  //calculate ctag
#ifdef TIME_MEASURE_DEF
  // start event
  hipEventRecord(start, 0);
#endif // TIME_MEASURE_DEF
  kernel_calc_ctag<<< grid, threads>>>( gpu_odata );
#ifdef TIME_MEASURE_DEF
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsedTime, start, stop);
  printf(" ::: kernel_calc_ctag time %f ms\n",elapsedTime);
#endif // TIME_MEASURE_DEF
#endif


#if 1
  // decimate the sum
#ifdef TIME_MEASURE_DEF
  // start event
  hipEventRecord(start, 0);
#endif // TIME_MEASURE_DEF
  int n_blocks_2 = HOST_N_samples/n_threads_per_block/HOST_decimation;
  dim3  grid_2( n_blocks_2, 1, 1);
  dim3  threads_2( n_threads_per_block, 1, 1);  
  kernel_decimate_sum<<< grid_2, threads_2>>>( gpu_odata );
#ifdef TIME_MEASURE_DEF
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsedTime, start, stop);
  printf(" ::: kernel_decimate_sum time %f ms\n",elapsedTime);
#endif // TIME_MEASURE_DEF
#endif

#if 1
  // fill sum all samples, all detectors
#ifdef TIME_MEASURE_DEF
  // start event
  hipEventRecord(start, 0);
#endif // TIME_MEASURE_DEF
  
  int n_blocks_3 = HOST_N_samples*N_DETECTORS/n_threads_per_block;
  dim3  grid_3( n_blocks_3, 1, 1);
  dim3  threads_3( n_threads_per_block, 1, 1);  
  kernel_wf_fillsum<<< grid_3, threads_3>>>( (ADC_TYPE*)gpu_idata, (ADC_TYPE*)gpu_odata );
#ifdef TIME_MEASURE_DEF
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsedTime, start, stop);
  printf(" ::: kernel_wf_fillsum time %f ms\n",elapsedTime);
#endif // TIME_MEASURE_DEF
#endif

  gpu_data_proc_size = 0;

#if 0
  // dummy copy. Used to wait for kernels
  int test;
  hipMemcpy( &test, gpu_odata + sizeof(GPU_AUX_DATA), sizeof(int), 
			     hipMemcpyDeviceToHost);
#endif    

#if 1
  // copy data from GPU
#ifdef TIME_MEASURE_DEF
  // start event
  hipEventRecord(start, 0);
#endif // TIME_MEASURE_DEF
  //ADC_TYPE test;
  //GPU_OUT_DATA data;
  //int data_size;

  /* tg fix to location of gpu_data_proc_size in gpu */
  hipMemcpy( &gpu_data_proc_size, gpu_odata + sizeof(GPU_HIS_DATA) + sizeof(GPU_AUX_DATA), sizeof(int), 
			     hipMemcpyDeviceToHost);
  //hipMemcpy( &gpu_data_proc_size, gpu_odata + sizeof(GPU_AUX_DATA), sizeof(int), 
  //			     hipMemcpyDeviceToHost);

  // include n_samples and data_size
  gpu_data_proc_size *= sizeof(int16_t); // size N islands with time stamp and island length
  // 32-bit island data size and 32-bit island counter
  gpu_data_proc_size += 2*sizeof(int);
  // 32-bit CTAG
  gpu_data_proc_size += 1*sizeof(int);

#ifdef TIME_MEASURE_DEF
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsedTime, start, stop);
  printf(" ::: copy data from GPU time %f ms\n",elapsedTime);
#endif // TIME_MEASURE_DEF

#if 1
  // add decimated histogram
  // We can also do this as modulo 24 in g-2 
  // but for testing I do for every fill
  gpu_data_proc_size += HOST_N_samples/HOST_decimation*sizeof(int16_t);
#endif

  dbprintf(" ::: GPU output data size %i\n",gpu_data_proc_size);
  //printf("result: %i\n",data.n_islands);

  if ( gpu_data_proc_size > gpu_data_proc_size_max )
    {
      printf("***ERROR! too large output gpu data! %i\n",gpu_data_proc_size);
      gpu_data_proc_size = 8;
    }

  hipMemcpy( cpu_odata, gpu_odata + sizeof(GPU_HIS_DATA)+ sizeof(GPU_AUX_DATA), gpu_data_proc_size, 
			     hipMemcpyDeviceToHost);
  //hipMemcpy( cpu_odata, gpu_odata + sizeof(GPU_AUX_DATA), gpu_data_proc_size, 
  //			     hipMemcpyDeviceToHost);
  
#endif

#ifdef TIME_MEASURE_DEF

  hipEventRecord(stop_all, 0);
  hipEventSynchronize(stop_all);
  hipEventElapsedTime(&elapsedTime, start_all, stop_all);
  dbprintf(" ::: CUDA kernel total elapsed time %f ms\n",elapsedTime);


  // Clean up:
  hipEventDestroy(start);
  hipEventDestroy(stop);
  hipEventDestroy(start_all);
  hipEventDestroy(stop_all);
#endif // TIME_MEASURE_DEF

  dbprintf(" ::: end-of-kernel, size of gpu_odata 0x%08x, GPU_HIS_DATA 0x%08x, GPU_AUX_DATA 0x%08x, GPU_OUT_DATA 0x%08x \n", 
  	   GPU_OBUF_SIZE, sizeof(GPU_HIS_DATA), sizeof(GPU_AUX_DATA), sizeof(GPU_OUT_DATA) );

}


#if 0
__global__
void kernel_wf_sum_make_islands( ADC_TYPE* gpu_idata, ADC* gpu_odata)
{
  // access thread id
  const unsigned int tid = threadIdx.x;
  // access number of threads in this block
  const unsigned int num_threads = blockDim.x;
  // access block id
  const unsigned int bid = blockIdx.x;
  // global index
  const unsigned int sample_nr = bid*num_threads + tid;

  CALORIMETER_DATA_BLOCK *cal_data = (CALORIMETER_DATA_BLOCK*) gpu_odata;

  // If sample is zero finish.
  // If sample is not zero, record 5 samples before and 10 after

  if ( sample_nr >= WAVEFORM_LENGTH_MAX ) return;
  
  if ( cal_data->wf_sum_thr.adc[sample_nr] == 0 ) return;
  
  if ( sample_nr > 0 && sample_nr < (WAVEFORM_LENGTH_MAX-1) )
    {
      if ( cal_data->wf_sum_thr.adc[sample_nr-1] != 0 &&  cal_data->wf_sum_thr.adc[sample_nr+1] != 0 )
	{
	  return;
	}
    }


  for (int i=1; i<7; i++)
    {    
      int s = sample_nr - i;
      if ( s>=0 )
	{
	  cal_data->wf_sum_thr.adc[s] = cal_data->wf_sum.adc[s];
	}
    }
  
  for (int i=1; i<24; i++)
    {    
      int s = sample_nr + i;
      if ( s < WAVEFORM_LENGTH_MAX )
	{
	  cal_data->wf_sum_thr.adc[s] = cal_data->wf_sum.adc[s];
	}
    }

}


__global__
void kernel_wf_sum_glue_islands(unsigned char* gpu_odata)
{

  CALORIMETER_DATA_BLOCK *calo = (CALORIMETER_DATA_BLOCK*)gpu_odata;
  int16_t *adc = calo->wf_sum_thr.adc;

  unsigned int i;
  bool sample_active = false;
  unsigned int N_islands = 0;
  //unsigned int island_len = 0;  
  unsigned int sample0 = 0;
  unsigned int offset = 0;
  // @todo replace WAVEFORM_LENGTH_MAX with actual wf length
  for (i=0; i<WAVEFORM_LENGTH_MAX; i++)
    {
      int16_t val = adc[i];
      if ( val == 0 )
	{
	  if ( sample_active )
	    {
	      // finish sample
	      calo->i_info[N_islands].sample0 = sample0;
	      unsigned int island_len = i - sample0;  
	      calo->i_info[N_islands].length = island_len;
	      N_islands++;
	      sample_active = false;
	      offset += ALIGN8(ISLAND_HEADER_LEN + island_len*ADC_SAMPLE_LEN);
	    }
	}
      else
	{
	  if ( sample_active )
	    {
	      // add new sample to the island
	      //calo->
	      ;
	    }
	  else
	    {
	      // start new island
	      calo->i_info[N_islands].offset = offset;
	      sample0 = i;
	      sample_active = true;
	    }
	}
    }
  
  calo->N_islands = N_islands;
  
  calo->Islands_len_total = offset;

}



__global__
void kernel_make_islands(unsigned char* gpu_odata)
{
  // access thread id
  const unsigned int tid = threadIdx.x;
  // access number of threads in this block
  const unsigned int num_threads = blockDim.x;
  // access block id
  const unsigned int bid = blockIdx.x;
  // global index
  const unsigned int island_nr = bid*num_threads + tid;

  CALORIMETER_DATA_BLOCK *cal_data = (CALORIMETER_DATA_BLOCK*) gpu_odata;

  unsigned int N_islands = cal_data->N_islands;

  if ( island_nr >= N_islands ) return;
  //if ( island_nr > 1 ) return;

  unsigned int sample0   = cal_data->i_info[island_nr].sample0;
  unsigned int len       = cal_data->i_info[island_nr].length;
  unsigned int offset    = cal_data->i_info[island_nr].offset;
  unsigned int len_total = cal_data->Islands_len_total;

  unsigned int iwf;
#if 0
  for (iwf=0; iwf<WAVEFORMS_NUM; iwf++)
#endif
#if 1
  for (iwf=0; iwf<1; iwf++)
#endif
    {
      /*
      unsigned char *ptr = (unsigned char*) &(cal_data->island);
      ISLAND_HEADER *island_header = (ISLAND_HEADER*)( ptr +  
						       iwf*len_total
						       + offset);
      */
      unsigned char *ptr = (unsigned char*) &(cal_data->island);
      int16_t *island = (int16_t*)( ptr + iwf*len_total + offset );
      //unsigned int *ptr_length  = ptr_sample0+1;
      //island_nr*(sizeof(ISLAND)-sizeof(unsigned short int)) + 
      //iwf*N_islands*(sizeof(ISLAND)-sizeof(unsigned short int));

      island[0] = sample0;
      island[1] = len;
      //unsigned short int *adc_tgt = (unsigned short int*)(island_header+1);
      //ptr = (unsigned char*)island_header; 
      //unsigned short int *adc_tgt = (unsigned short int*)(ptr+sizeof(ISLAND_HEADER));
      unsigned short int *adc_tgt = (unsigned short int*)(island+2);
      unsigned short int *adc_src = cal_data->wf[iwf].adc;
      //cal_data->wf_sum.adc[sample_nr] += cal_data->wf[i].adc[sample_nr];
      unsigned int i;
      for (i=0; i<len; i++)
	{
	  adc_tgt[i] = adc_src[sample0+i]; 
	  //adc_tgt[i] = i+1;//adc_src[sample0+i]; 
	}

#if 0
      island_header->length = 10;
      island_header->sample0 = 20;
#endif

    }


}

#endif







